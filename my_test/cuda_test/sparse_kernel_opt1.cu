#include "hip/hip_runtime.h"
#include <torch/extension.h>

__global__ void sparse_scatter_kernel_improved(
    const int* __restrict__ indices,    // [4, N]
    const double* __restrict__ values,  // [N]
    const double* __restrict__ B,       // [B0, B1, B2]
    double* __restrict__ output,        // [B0, B1, 16,16,16]
    int num_nonzeros,
    int B0, int B1, int B2
) {
    int n = blockIdx.z;
    int i = blockIdx.x * blockDim.x + threadIdx.x; // B0
    int j = blockIdx.y * blockDim.y + threadIdx.y; // B1

    if (n >= num_nonzeros || i >= B0 || j >= B1) return;

    int i0 = indices[n];
    int i1 = indices[n + num_nonzeros];
    int i2 = indices[n + 2 * num_nonzeros];
    int i3 = indices[n + 3 * num_nonzeros];
    double a_val = values[n];

    double b_val = B[i * B1 * B2 + j * B2 + i3];
    double prod = a_val * b_val;

    int out_idx = i * B1 * 16 * 16 * 16 + j * 16 * 16 * 16 + i0 * 16 * 16 + i1 * 16 + i2;

    atomicAdd(&output[out_idx], prod);
}

void sparse_scatter_mul_add_improved(
    at::Tensor indices,
    at::Tensor values,
    at::Tensor B,
    at::Tensor output
) {
    int num_nonzeros = values.size(0);
    int B0 = B.size(0);
    int B1 = B.size(1);
    int B2 = B.size(2);

    dim3 threads(32, 8);
    dim3 blocks(
        (B0 + threads.x - 1) / threads.x,
        (B1 + threads.y - 1) / threads.y,
        num_nonzeros
    );

    sparse_scatter_kernel_improved<<<blocks, threads>>>(
        indices.data_ptr<int>(),
        values.data_ptr<double>(),
        B.data_ptr<double>(),
        output.data_ptr<double>(),
        num_nonzeros, B0, B1, B2
    );
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("sparse_scatter_mul_add_optimized", &sparse_scatter_mul_add_improved, "Optimized sparse scatter multiply-add");
}
